//Just your regular Hello World file
// to be compiled with nvcc rather than gcc


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
  printf("Hello World from GPU, thread %d\n",threadIdx.x);
}


int main(void) {
  printf("Hello World from CPU!\n");

  helloFromGPU<<<1, 10>>>();
  hipDeviceReset();

  return 0;
}
