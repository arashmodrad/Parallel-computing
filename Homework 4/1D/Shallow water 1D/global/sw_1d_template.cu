#include "hip/hip_runtime.h"
# include <stdlib.h>
# include <stdio.h>
# include <math.h>
# include <string.h>
# include <time.h>
# include "common.h"




int main ( int argc, char *argv[] );
void initial_conditions ( int nx, double dx, double x_length, double x[], double h[], double uh[]);

//utilities
void getArgs(int *nx, double *dt, double *x_length, double *t_final, int *THREADS_PER_BLOCK, int argc, char *argv[]);
void write_results ( char *output_filename, int n, double x[], double h[], double uh[]);

__global__ void shallow_water_kernel(double *d_h, double *d_uh, double *fh, double *fuh,double *hm, double *uhm, double lambda, int nx, double g)
{
  //declare global index
  int gidx = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int i, j;
  
  // K1) Compute interior fluxes - switch a loop for an if statement
  //for ( i = 1; i < nx+1; i++ )
  if(gidx<nx+1) // want to do only interior nodes
    {
      fh[gidx] = d_uh[gidx]; //flux for the height equation: u*h
      fuh[gidx] = d_uh[gidx]*d_uh[gidx]/d_h[gidx] + 0.5*g*d_h[gidx]*d_h[gidx]; //flux for the momentum equation: u^2*h + 0.5*g*h^2
    }

  // K2) Compute ghost fluxes (need ghost values) - switch loops for if statements
  //left ghost 
  if(gidx==1){
    i=0;
    fh[i] = d_uh[i];
    fuh[i] = d_uh[i]*d_uh[i]/d_h[i] + 0.5*g*d_h[i]*d_h[i];
  }
  if(gidx==nx){ 
    //right ghost
    j = nx+1;
    fh[j] = d_uh[j];
    fuh[j] = d_uh[j]*d_uh[j]/d_h[j] + 0.5*g*d_h[j]*d_h[j];
  }

  // K3) synchronize threads
  __syncthreads();
  
  // K4) Compute updated variables - swtich a loop for an if statement
  // switch i index to global index, or store gidx as i if you want to save typing
  if (gidx > 0 && gidx < nx + 1)
  {
	  i = gidx;
	  hm[i] = 0.5*(d_h[i + 1] + d_h[i - 1]) - lambda * (fh[i + 1] - fh[i - 1]);
	  uhm[i] = 0.5*(d_uh[i + 1] + d_uh[i - 1]) - lambda * (fuh[i + 1] - fuh[i - 1]);
  }
  /*for ( i = 1; i < nx; i++ )
    {
      hm[i] = 0.5*(d_h[i+1]+d_h[i-1]) - lambda * ( fh[i+1] - fh[i-1] );
      uhm[i] = 0.5*(d_uh[i+1]+d_uh[i-1]) - lambda * ( fuh[i+1] - fuh[i-1] );
    }
	*/
  // K5) Update the boundary conditions - only first and last thread (0 and nx)
  // will have to do this
  if (gidx == 1)
  {
	  hm[0] = hm[1];
	  uhm[0] = -uhm[1];
  }
  if (gidx == nx)
  {
	  hm[nx + 1] = hm[nx];
	  uhm[nx + 1] = -uhm[nx];
  }
  
  
  // K6) Synchronize threads to make sure updates are computed everywhere
  __syncthreads();

  // K7) Update state variables - put the value from hm and uhm to solution arrays
  // replace loop with an if
  if (gidx > 0 && gidx < nx + 1)
  {
	  i = gidx;
	  d_h[i] = hm[i];
	  d_uh[i] = uhm[i];
  }
  /*for (i = 1; i < nx+1; i++){
    d_h[i] = hm[i];
    d_uh[i] = uhm[i];
  }*/

}

/******************************************************************************/

int main ( int argc, char *argv[] )

/******************************************************************************/
/*
  Purpose:
    MAIN is the main program for SHALLOW_WATER_1D.

  Discussion:
    SHALLOW_WATER_1D approximates the 1D shallow water equations.
    The version of the shallow water equations being solved here is in
    conservative form, and omits the Coriolis force.  The state variables
    are H (the height) and UH (the mass velocity).

    The equations have the form
      dH/dt + d UH/dx = 0
      d UH/dt + d ( U^2 H + 1/2 g H^2 )/dx = 0

    Here U is the ordinary velocity, U = UH/H, and g is the gravitational
    acceleration.
    The initial conditions are used to specify ( H, UH ) at an equally
    spaced set of points, and then the Lax-Friedrichs method is used to advance
    the solution until a final time t_final, with
    boundary conditions supplying the first and last spatial values.
    Some input values will result in an unstable calculation that
    quickly blows up.  This is related to the Courant-Friedrichs-Levy
    condition, which requires that DT be small enough, relative to DX and
    the velocity, that information cannot cross an entire cell.

    A "reasonable" set of input quantities is
      shallow_water_1d 401 0.002 10.0 0.2

  Licensing:
    This code is distributed under the GNU LGPL license.

  Modified:
    26 March 2019 by Michal A. Kopera

  Parameters:
    Input, integer NX, the number of spatial nodes.
    Input, integer DT, the size of a time step.
    Input, real X_LENGTH, the length of the region.
    Input, real T_FINAL, the final time of simulation.

    Output, real X[NX], the X coordinates.
    Output, real H[NX], the height for all space points at time t_final.
    Output, real UH[NX], the mass velocity (discharge) for all space points at time t_final.
*/
{
printf("%s Starting...\n", argv[0]);

// set up device
int dev = 0;
hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
  int THREADS_PER_BLOCK;
  double dx;
  double dt;
  double g = 9.81; //[m^2/s] gravitational constant
  double *h;
  double *fh;
  double *hm;
  int nx;
  double t_final;
  double *uh;
  double *fuh;
  double *uhm;
  double *x;
  double x_length, time;


printf ( "\n" );
printf ( "SHALLOW_WATER_1D\n" );
printf ( "\n" );


  //get command line arguments
getArgs(&nx, &dt, &x_length, &t_final, &THREADS_PER_BLOCK, argc, argv);

    printf ( "  NX = %d\n", nx );
    printf ( "  DT = %g\n", dt );
    printf ( "  X_LENGTH = %g\n", x_length );
    printf ( "  T_FINAL = %g\n", t_final );
	printf("  THREADS_PER_BLOCK = %d\n", THREADS_PER_BLOCK);

  //M1) Allocate space (nx+2) long, to accound for ghosts
  //height array
    size_t nBytes = (nx+2)*sizeof(double);
  h = ( double * ) malloc ( nBytes );
  //discharge array
  uh = ( double * ) malloc ( nBytes);
  // location array
  x = ( double * ) malloc ( nx * sizeof ( double ) );

  //Define the locations of the nodes and time steps and the spacing.
  dx = x_length / ( double ) ( nx );

  // M2) Apply the initial conditions.
  initial_conditions ( nx, dx, x_length,  x, h, uh);

  // M3) Write initial condition to a file
  write_results((char *)"sw1d_cuda_init.dat",nx,x,h,uh);

  double lambda = 0.5*dt/dx;

  // M4) allocate device memory to hold h, uh, hm, uhm, fh, fuh
  // some memory will be moved between device and host (h, uh), so
  // I encourage you to use d_h and d_uh notation for those variables
  // variables which live only on the device do not need that
  // In principle, they can be allocated from the device, but since we
  // need to allocate them only once, I suggest to do it from the host
  // Use hipMalloc and nBytes to allocate d_h, d_uh, hm, uhm, fh, fuh
  double *d_h, *d_uh;
  CHECK(hipMalloc((double **)&d_h, nBytes));
  CHECK(hipMalloc((double **)&d_uh, nBytes));
  CHECK(hipMalloc((double **)&hm, nBytes));
  CHECK(hipMalloc((double **)&uhm, nBytes));
  CHECK(hipMalloc((double **)&fh, nBytes));
  CHECK(hipMalloc((double **)&fuh, nBytes));

  // M5) transfer data from host to device
  // use hipMemcpy to transfer h and uh to device
  CHECK(hipMemcpy(d_h, h, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_uh, uh, nBytes, hipMemcpyHostToDevice));

  // M6) initialize kernel data
  int nThreads = THREADS_PER_BLOCK;
  int nBlocks =  ((nx + nThreads - 1) / nThreads);

  //start timer
  double iStart, iElaps;
  iStart = seconds();

  time=0;
  while (time<t_final) //for ( it = 1; it <= nt; it++ )
    {
      //  Take a time step
      time=time+dt;
      //printf("time = %f\n",time);

      // M7)  call computational kerne
      shallow_water_kernel<<<nBlocks,nThreads>>>(d_h, d_uh, fh, fuh, hm, uhm, lambda, nx, g);
      
      // check kernel error
      CHECK(hipDeviceSynchronize());
      CHECK(hipGetLastError());


      // PARALLELIZATION ENDS HERE
    }

  CHECK(hipDeviceSynchronize());
  iElaps = seconds() - iStart;
  printf("shallow_water <<<  %d, %d  >>>  Time elapsed %f sec\n", nBlocks,
	 nThreads, iElaps);
    
  // M8) copy kernel result back to host side
  // use hipMemcpy to get values from d_h and d_uh to the host
  CHECK(hipMemcpy(h, d_h, nBytes, hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(uh, d_uh, nBytes, hipMemcpyDeviceToHost));

  // M9) Write data to file
  write_results((char *)"sw1d_cuda_final.dat",nx,x,h,uh);
  
  // M10) Free host memory.
  free ( h );
  free ( uh );
  free ( x );

  // M11) Free device memory using hipFree


 //Terminate.
  printf ( "\n" );
  printf ( "SHALLOW_WATER_1D:\n" );
  printf ( "  Normal end of execution.\n" );
  printf ( "\n" );

  return 0;
}
/******************************************************************************/

void initial_conditions ( int nx, double dx, double x_length, double x[], double h[], 
			  double uh[])

/******************************************************************************/

{
  int i;
  
  for ( i = 1; i < nx+1; i++ )
    {
      x[i-1] = -x_length/2+dx/2+(i-1)*dx;
      double xx = x[i-1];
      h[i] = 1.0 + 0.4*exp ( -5 * ( xx*xx) );
    }
  for ( i = 1; i < nx+1; i++ )
    {
      uh[i] = 0.0;
    }
  h[0] = h[1];
  h[nx+1]=h[nx];
  uh[0] = 0.0;
  uh[nx+1] = 0.0;
  return;
}
/******************************************************************************/


void write_results ( char *output_filename, int n, double x[], double h[], double uh[])
/******************************************************************************/

{
  int j;
  FILE *output;
  
  //Open the file.
  output = fopen ( output_filename, "wt" );
    
  if ( !output ){
    fprintf ( stderr, "\n" );
    fprintf ( stderr, "WRITE_RESULTS - Fatal error!\n" );
    fprintf ( stderr, "  Could not open the output file.\n" );
    exit ( 1 );
  }
    
  //Write the data.
  for ( j = 1; j < n+1; j++ )	{
    fprintf ( output, "  %24.16g\t %24.16g\t %24.16g\n", x[j-1], h[j], uh[j]);
  }
  
  //Close the file.
  fclose ( output );
  return;
}
/******************************************************************************/

void getArgs(int *nx, double *dt, double *x_length, double *t_final, int *THREADS_PER_BLOCK, int argc, char *argv[])
{

    /*
      Get the quadrature file root name:
    */
    if ( argc <= 1 ){
      *nx = 401;
    }else{
      *nx = atoi ( argv[1] );
    }
    
    if ( argc <= 2 ){
      *dt = 0.002;
    }else{
      *dt = atof ( argv[2] );
    }
    
    if ( argc <= 3 ){
      *x_length = 10.0;
    }else{
      *x_length = atof ( argv[3] );
    }
    
    if ( argc <= 4 ){
      *t_final = 0.5;
    }else{
      *t_final = atof ( argv[4] );
    }
	if (argc <= 5){
		*THREADS_PER_BLOCK = 32;
	}
	else{
		*THREADS_PER_BLOCK = atof(argv[5]);
	}
  
}
